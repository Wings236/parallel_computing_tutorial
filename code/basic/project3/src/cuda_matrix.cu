#include "hip/hip_runtime.h"
#include "matrix.h"
#include <hip/hip_runtime.h>

const int threadDim = 32;
const int TILE_SIZE = threadDim;
__global__ void matrixMul_blockC(float *A, float *B, float *C, int A_row, int A_column, int B_column){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	float acc = 0;
	for (int k = 0; k < A_column; k++)
		acc += A[j * A_column + k] * B[k * B_column + i];
	C[j * B_column +i] = acc;
}

__global__ void matrixMultiplyShared(const float *A, const float *B, float *C, int A_row, int A_column, int B_column) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float As[TILE_SIZE][TILE_SIZE];
	__shared__ float Bs[TILE_SIZE][TILE_SIZE];

	float value = 0;

	for (int i = 0; i < A_column / TILE_SIZE; i++){
		As[threadIdx.y][threadIdx.x] = A[(blockIdx.y * TILE_SIZE + threadIdx.y) * A_column + TILE_SIZE * i + threadIdx.x];
		Bs[threadIdx.y][threadIdx.x] = B[(i * TILE_SIZE + threadIdx.y) * B_column + blockIdx.x * TILE_SIZE + threadIdx.x];

		__syncthreads();

		for (int k = 0; k < TILE_SIZE; k++)
			value += As[threadIdx.y][k] * Bs[k][threadIdx.x];

		__syncthreads();
	}


	C[row * B_column + col] = value;
}



int cuda_matmulMatrix(const Matrix *MatA, const Matrix *MatB, Matrix *MatC) {
    if(MatA->ROW_NUM != MatC->ROW_NUM || MatA->COL_NUM != MatB->ROW_NUM || MatB->COL_NUM != MatC->ROW_NUM) return -1;
    size_t rowA = MatA->ROW_NUM;
    size_t colA = MatA->COL_NUM;
    size_t rowB = MatB->ROW_NUM;
    size_t colB = MatB->COL_NUM;

    // CUDA section
    float *d_A;
    float *d_B;
    float *d_C;

    // Allocate memory
    hipMalloc(&d_A, rowA*colA*sizeof(float));
    hipMalloc(&d_B, rowB*colB*sizeof(float));
    hipMalloc(&d_C, rowA*colB*sizeof(float));

    // Copy data to GPU
    hipMemcpy(d_A, MatA->data, rowA*colA*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, MatB->data, rowB*colB*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, MatC->data, rowA*colB*sizeof(float), hipMemcpyHostToDevice);

    const dim3 threadsPerBlock(threadDim, threadDim);
    const dim3 numBlocks(colB / threadsPerBlock.x, rowA / threadsPerBlock.y);
    // Invoke the cuda imp.
    matrixMultiplyShared<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, rowA, colA, colB);

    // Get the result back
    hipMemcpy(MatC->data, d_C, rowA*colB*sizeof(float), hipMemcpyDeviceToHost);
    return 0;
}
