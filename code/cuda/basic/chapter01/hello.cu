
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void)
{
    printf("Hello World from GPU!\n");
}

int main()
{
    // CPU
    printf("Hello World from CPU!\n");

    // GPU
    helloFromGPU<<<1, 10>>>();
    hipDeviceReset();
    return 0;
}