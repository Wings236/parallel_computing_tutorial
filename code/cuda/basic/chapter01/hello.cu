
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void)
{

    if(threadIdx.x == 5)
    {
        printf("Hello World from GPU thread 5!\n");
    }

}

int main()
{
    // CPU
    printf("Hello World from CPU!\n");

    // GPU
    helloFromGPU<<<1, 10>>>();
    hipDeviceReset();
    // cudaDeviceSynchronize();
    return 0;
}